
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstring>
#include <iostream>
#include <vector>
#include <chrono>

const uint64_t MAX_STRIDE = 256000000;
int BLOCK_SIZE = 256;
bool* sieve_buffer_host = nullptr;

__global__ void sieve_kernel(uint64_t max, bool *sieve_buffer)
{
    uint64_t index = blockIdx.x * blockDim.x + threadIdx.x; // index = range of 0 up to MAX_STRIDE
    uint64_t stride = blockDim.x * gridDim.x;               // stride is a number of total threads in grid =>  
                                                            // Grid contains X number of Blocks which contains Y number of Threads 
    for (uint64_t i = index; i <= max; i += stride)
    {
        if(sieve_buffer[i]) {
            uint64_t j = i;
            uint64_t val = i + j + 2 * i * j;
            while (val <= max && val >= i + j)
            {
                uint64_t index2 = i + j + 2 * i * j;
                sieve_buffer[index2] = false;
                j += 1;
                val = i + j + 2 * i * j;
            }
        }
    }
}

void sieve_of_sundaram_gpu_followup(int max)
{
    bool *sieve_buffer_device = nullptr;
    // uint64_t *seed_primes_device = nullptr;

    // allocate on cpu and gpu array of bool and set default true
    sieve_buffer_host = (bool*) malloc(max * sizeof(bool));
    hipMalloc(&sieve_buffer_device, max * sizeof(bool));
    std::memset(sieve_buffer_host, true, max * sizeof(bool));
    hipMemcpy(sieve_buffer_device, sieve_buffer_host, max * sizeof(bool), hipMemcpyHostToDevice);

    // allocate on cpu and gpu for seed_primes and copy seed_primes to gpu
    // uint64_t seed_primes_size = seed_primes.size();
    // uint64_t *seed_primes_host = (uint64_t *)malloc(seed_primes_size * sizeof(uint64_t));
    // memcpy(seed_primes_host, seed_primes.data(), seed_primes_size * sizeof(uint64_t));
    // cudaMalloc(&seed_primes_device, seed_primes_size * sizeof(uint64_t));
    // cudaMemcpy(seed_primes_device, seed_primes_host, seed_primes_size * sizeof(uint64_t), cudaMemcpyHostToDevice);

    // calculate number of blocks
    uint64_t num_blocks = (max + BLOCK_SIZE - 1) / BLOCK_SIZE;
    uint64_t stride = BLOCK_SIZE * num_blocks;

    if (stride > MAX_STRIDE)
    {
        num_blocks = MAX_STRIDE / BLOCK_SIZE;
    }

    // run on GPU
    sieve_kernel<<<num_blocks, BLOCK_SIZE>>>(max, sieve_buffer_device);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();
    hipMemcpy(sieve_buffer_host, sieve_buffer_device, max * sizeof(bool), hipMemcpyDeviceToHost);

    // free(seed_primes_host);
    hipFree(sieve_buffer_device);
    // cudaFree(seed_primes_device);
}

void check_primes(int target)
{

    uint64_t count = 1;
    // std::cout << 2 << " ";
    for (int p = 1; p <= target; p++)
    {
        if (sieve_buffer_host[p])
        {
            count++;
            // std::cout << 2 * p + 1 << " ";
        }
    }
    std::cout << "Found " << count << " prime numbers" << std::endl; 
}

int main(int argc, char *argv[])
{
    int target = 400;

    for (int i = 1; i < argc; ++i) {
        int integerValue = std::atoi(argv[i]);

        if (integerValue == 0 && argv[i][0] != '0') {
            std::cerr << "Invalid integer: " << argv[i] << std::endl;
            
        }else{
            if(i==1){
                target = integerValue;
            }
            if(i==2){
                BLOCK_SIZE = integerValue;
            }
        }
    }

    auto start = std::chrono::high_resolution_clock::now();
    int k = (target - 2) / 2;
    // std::vector<uint64_t> seed_primes = sieve_of_sundaram_cpu(sqrt);
    sieve_of_sundaram_gpu_followup(k+1);

    auto end = std::chrono::high_resolution_clock::now();

    auto duration = std::chrono::duration_cast<std::chrono::microseconds>(end - start);

    std::cout << "Time taken by threads: "
              << duration.count() << " microseconds" << std::endl;

    check_primes(k);
}